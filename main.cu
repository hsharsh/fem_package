#include "hip/hip_runtime.h"
#include "structure.h"
#include "assembly.h"
#include "device.cuh"
#include <bits/stdc++.h>
using namespace std;

int main(int argc, char** argv){
	freopen("paralleloutput.o","w",stdout);
	ll nelm = 0, tnod = 0, ndof = 0, nnod = 0;
	string type;

	ifstream input(argv[1]);
	vdd connectivity;
	vdd x;

	for(string line; getline(input,line);){
		if(line.substr(0,5) == "*Node"){
			while(getline(input, line) && (line.substr(0,1).compare("*"))){
				stringstream ss(line);
				string s;
				vd temp;
				REP(j, 4){
					ss >> s;

					int decimal = s.size()-1;
					REP(k, s.size()){
						if(s[k] == '.')
							decimal = k;
					}

					double num = 0;
					
					bool negative = false;
					if(s.substr(0,1) == "-")
						negative = true;
					num = stoll(s.substr((negative ? 1 : 0),decimal - (negative ? 1 : 0)));
					if(decimal != (s.size()-1)){
						num += stoll("0" + s.substr(decimal+1,s.size()-decimal-2)) * pow(10, -1 * (int)(s.size()-decimal-2)) ;
					}
					if(negative)
						num *= -1;
					
					if(j != 0)
						temp.push_back(num);
					ss.ignore();
				}
				x.push_back(temp);
				tnod++;

			}
		}

		if(line.substr(0,8) == "*Element"){

			type = line.substr(15,line.size()-15);
 			if(type.size() >= 4 && type.substr(0,4) == "C3D4"){
				ndof = 3;
				nnod = 4;
			}
			if(type.size() >= 5 && type.substr(0,5) == "C3D10"){
				ndof = 3;
				nnod = 10;
			}
			cout << nnod << endl;
			while(getline(input, line) && (line.substr(0,1).compare("*"))){
				cout << line << endl;
				stringstream ss(line);
				ll k;
				vd temp;

				REP(j, nnod + 1){
					ss >> k;
					if(j != 0)
						temp.push_back(k);
					ss.ignore();
				}
				nelm++;
				connectivity.push_back(temp);
			}

		}
	}


	Assembly 		m_assembly(nelm, tnod, ndof);
	vector<Node> 	nodes(tnod,Node(ndof));
	if(type.size() >= 4 && type.substr(0,4) == "C3D4")
		vector<Element> mesh(nelm,Tetra_linear());
	if(type.size() >= 5 && type.substr(0,5) == "C3D10")
		vector<Element> mesh(nelm,Tetra_quadratic());

	REP(i, tnod){
		nodes[i].build_x(x[i]);
	}


	cout << "Nodes: " << endl << endl;
	REP(i, tnod){
		cout << "Node " << i+1 << ": ";
		REP(j, 3){
			cout << x[i][j] << " ";
		}
		cout << endl;
	}

	double kl[nelm*nnod*ndof*nnod*ndof];

	REP(k, nelm){
		REP(i, nnod*ndof){
			REP(j, nnod*ndof){
				kl[k*nnod*ndof*nnod*ndof + i*nnod*ndof + j] = 0.0;
				//cout << kl[i*nnod*ndof + j] << " ";
			}
			//cout << endl;
		}
	}

	cout << endl;
	cout << "Connectivity matrix: " << endl << endl;
	REP(i,nelm){
		cout << "Element " << i+1 << ": ";
		vdd node(nnod,vd(3));
	    REP(j, nnod){
	        REP(k, 3){
	        	//cout << connectivity[i][j] << "->" << x[connectivity[i][j]-1][k] << " ";
	            node[j][k] = x[connectivity[i][j]-1][k];
	        }
	       // cout << endl;
	    }
	    vdd k_local;
	    if(type.size() >= 4 && type.substr(0,4) == "C3D4")
			k_local = build_k(node);
		if(type.size() >= 5 && type.substr(0,5) == "C3D10")
			k_local = vdd(30,vd(30,0));
		
		REP(j, nnod*ndof){
			REP(k, nnod*ndof){
				kl[i*nnod*ndof*nnod*ndof + j*nnod*ndof + k] = k_local[j][k];
				//cout << kl[i*nnod*ndof + j] << " ";
			}
			//cout << endl;
		}

		REP(j,nnod)
			cout << connectivity[i][j]<< " ";
		cout << endl;
	}
	m_assembly.build_connectivity(connectivity);
/*
	cout << "Connectivity matrix : " << endl;
	REP(i, nelm){
		for(auto j : m_assembly.connectivity[i]){
			cout << j << " ";
		}
		cout << endl;
	}
*/

	int dof[nelm*nnod*ndof];
	REP(i, nelm){
		REP(j, m_assembly.connectivity[i].size()){
			REP(k, ndof){
				dof[i*nnod*ndof + j*ndof + k] = (ndof * m_assembly.connectivity[i][j]) - (ndof - k);
				//cout << dof[i*nnod*ndof + j*ndof + k] << " ";
			}
		}
		//cout << endl;
	}

	double kg[tnod*ndof*tnod*ndof];
	
	REP(i, tnod*ndof){
		REP(j, tnod*ndof){
			kg[i*tnod*ndof + j] = 0.0;
		}
	}


/*
	cout << endl << "Dof matrix:" << endl;
	REP(i, nelm){
		REP(j, nnod*ndof){
			cout << dof[i*nnod*ndof + j] << " ";
		}
		cout << endl;
	}
*/

	double (*pkg), (*pkl);
	int (*pdof);

	hipMalloc((void**)&pdof, (nelm*nnod*ndof)*sizeof(int));
	hipMalloc((void**)&pkg, (tnod*ndof*tnod*ndof)*sizeof(double));
	hipMalloc((void**)&pkl, (nelm*nnod*ndof*nnod*ndof)*sizeof(double));

	hipMemcpy(pdof, dof, (nelm*nnod*ndof)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pkl, kl, (nelm*nnod*ndof*nnod*ndof)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(pkg, kg, (tnod*ndof*tnod*ndof)*sizeof(double), hipMemcpyHostToDevice);

	int numBlocks = nelm;
	dim3 threadsPerBlock(nnod*ndof,nnod*ndof);
	MatAdd<<<numBlocks,threadsPerBlock>>>(pkg,pkl,pdof,nnod*ndof,tnod*ndof);

	hipMemcpy(kg, pkg, (tnod*ndof*tnod*ndof)*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(pdof); 
	hipFree(pkl);
	hipFree(pkg);


	cout << endl;
	cout << "K global:" << endl << endl;
	REP(i,tnod*ndof){
		REP(j,tnod*ndof){
			cout << kg[i*tnod*ndof + j] << " ";
		}
		cout << ";" << endl;
	}
}
