#include "hip/hip_runtime.h"
#include "structure.h"
#include <bits/stdc++.h>
using namespace std;


__device__ double datomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void MatAdd(double kg[], double kl[], int dof[], int sizekl, int sizekg){
           int i = blockIdx.x;
           int j = threadIdx.x;
           int k = threadIdx.y;
           //printf("Element %d: %d %d Dofi:%d Dofj:%d Kl:%d Kg:%d\n", i, j, k, dof[i][j], dof[i][k], kl[j][k], kg[dof[i][j]][dof[i][k]]);
           //kg[dof[i][j]][dof[i][k]] = kg[dof[i][j]][dof[i][k]] + kl[j][k];
           datomicAdd( &(kg[dof[i*sizekl + j]*sizekg + dof[i*sizekl + k]]), kl[j*sizekl +k]);
			__syncthreads();
       }

ll stoll(string s){
	ll number = 0;
	DFOR(i,s.size()-1,0){
		number = number + (s[i]-'0') * pow(10,s.size()-i-1);
	}
	return number;
}

int main(int argc, char** argv){
	freopen("paralleloutput.o","w",stdout);
	ll nelm = 0, tnod = 0, ndof = 0, nnod = 0;
	string type;

	ifstream input(argv[1]);
	vdd connectivity;
	vdd x;

	for(string line; getline(input,line);){
		if(line.substr(0,5) == "*Node"){
			while(getline(input, line) && (line.substr(0,1).compare("*"))){
				stringstream ss(line);
				string s;
				vd temp;
				REP(j, 4){
					ss >> s;

					int decimal = s.size()-1;
					REP(k, s.size()){
						if(s[k] == '.')
							decimal = k;
					}

					double num = 0;
					num = stoll(s.substr(0,decimal));
					if(decimal != (s.size()-1)){
						num += stoll("0" + s.substr(decimal+1,s.size()-decimal-2)) * pow(10, -1 * (int)(s.size()-decimal-2)) ;
					}
					if(j != 0)
						temp.push_back(num);
					ss.ignore();
				}
				x.push_back(temp);
				tnod++;

			}
		}

		if(line.substr(0,8) == "*Element"){
			type = line.substr(15,line.size()-15);
			if(type == "C3D4"){
				ndof = 3;
				nnod = 4;
			}
			while(getline(input, line) && (line.substr(0,1).compare("*"))){
				stringstream ss(line);
				ll k;
				vd temp;
				REP(j, nnod + 1){
					ss >> k;
					if(j != 0)
						temp.push_back(k);
					ss.ignore();
				}
				nelm++;				
				connectivity.push_back(temp);
			}
		}
	}


	Assembly 		m_assembly(nelm, tnod, ndof);
	vector<Node> 	nodes(tnod,Node(ndof));
	if(type == "C3D4")
		vector<Element> mesh(nelm,Tetra());

	REP(i, tnod){
		nodes[i].build_x(x[i]);
	}


	cout << "Nodes: " << endl << endl;
	REP(i, tnod){
		cout << "Node " << i+1 << ": ";
		REP(j, 3){
			cout << x[i][j] << " ";
		}
		cout << endl;
	}


	cout << endl;
	cout << "Connectivity matrix: " << endl << endl;
	REP(i,nelm){
		cout << "Element " << i+1 << ": ";
		REP(j,nnod)
			cout << connectivity[i][j]<< " ";
		cout << endl;
	}
	m_assembly.build_connectivity(connectivity);
/*
	cout << "Connectivity matrix : " << endl;
	REP(i, nelm){
		for(auto j : m_assembly.connectivity[i]){
			cout << j << " ";
		}
		cout << endl;
	}
*/

	int dof[nelm*nnod*ndof];
	REP(i, nelm){
		REP(j, m_assembly.connectivity[i].size()){
			REP(k, ndof){
				dof[i*nnod*ndof + j*ndof + k] = (ndof * m_assembly.connectivity[i][j]) - (ndof - k);
			}
		}
	}

	double kg[tnod*ndof*tnod*ndof];
	
	REP(i, tnod*ndof){
		REP(j, tnod*ndof){
			kg[i*tnod*ndof + j] = 0.0;
		}
	}

	double kl[nnod*ndof*nnod*ndof];

	REP(i, nnod*ndof){
		REP(j, nnod*ndof){
			kl[i*nnod*ndof + j] = 1.0;
		}
	}	
/*
	cout << endl << "Dof matrix:" << endl;
	REP(i, nelm){
		REP(j, nnod*ndof){
			cout << dof[i*nnod*ndof + j] << " ";
		}
		cout << endl;
	}
*/

	double (*pkg), (*pkl);
	int (*pdof);

	hipMalloc((void**)&pdof, (nelm*nnod*ndof)*sizeof(int));
	hipMalloc((void**)&pkg, (tnod*ndof*tnod*ndof)*sizeof(double));
	hipMalloc((void**)&pkl, (nnod*ndof*nnod*ndof)*sizeof(double));

	hipMemcpy(pdof, dof, (nelm*nnod*ndof)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pkl, kl, (nnod*ndof*nnod*ndof)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(pkg, kg, (tnod*ndof*tnod*ndof)*sizeof(double), hipMemcpyHostToDevice);

	int numBlocks = nelm;
	dim3 threadsPerBlock(nnod*ndof,nnod*ndof);
	MatAdd<<<numBlocks,threadsPerBlock>>>(pkg,pkl,pdof,nnod*ndof,tnod*ndof);

	hipMemcpy(kg, pkg, (tnod*ndof*tnod*ndof)*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(pdof); 
	hipFree(pkl);
	hipFree(pkg);


	cout << endl;
	cout << "K global:" << endl << endl;
	REP(i,tnod*ndof){
		REP(j,tnod*ndof){
			cout << kg[i*tnod*ndof + j] << " ";
		}
		cout << endl;
	}
}
