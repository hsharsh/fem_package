#include "hip/hip_runtime.h"
#include "device.cuh"

__device__ double datomicAdd(double* address, double val){
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do{
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while(assumed != old);

    return __longlong_as_double(old);
}

__global__ void MatAdd(double kg[], double kl[], int dof[], int sizekl, int sizekg){
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k = threadIdx.y;
    datomicAdd( &(kg[dof[i*sizekl + j]*sizekg + dof[i*sizekl + k]]), kl[i*sizekl*sizekl + j*sizekl +k]);
    __syncthreads();
}
